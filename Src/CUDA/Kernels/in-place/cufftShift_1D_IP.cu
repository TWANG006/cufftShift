#include "hip/hip_runtime.h"
#ifndef CUFFTSHIFT_1D_SINGLE_ARRAY_CU
#define CUFFTSHIFT_1D_SINGLE_ARRAY_CU

#include <hip/hip_runtime.h>
#include <cutil_inline.h>

template <typename T>
__global__
void cufftShift_1D_kernel(T* data, int NX)
{
    int threadIdxX = threadIdx.x;
    int blockDimX = blockDim.x;
    int blockIdxX = blockIdx.x;

    int index = ((blockIdxX * blockDimX) + threadIdxX);
    if (index < NX/2)
    {
        // Save the first value
        T regTemp = data[index];

        // Swap the first element
        data[index] = (T) data[index + (NX / 2)];

        // Swap the second one
        data[index + (NX / 2)] = (T) regTemp;
    }
}

#endif // CUFFTSHIFT_1D_SINGLE_ARRAY_CU
