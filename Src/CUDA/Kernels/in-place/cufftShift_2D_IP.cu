#include "hip/hip_runtime.h"
#ifndef CUFFTSHIFT_2D_SINGLE_ARRAY_CU
#define CUFFTSHIFT_2D_SINGLE_ARRAY_CU

#include <hip/hip_runtime.h>
#include <cutil_inline.h>

template <typename T>
__global__
void cufftShift_2D_kernel(T* data, int N)
{
    // 2D Slice & 1D Line
    int sLine = N;
    int sSlice = N * N;

    // Transformations Equations
    int sEq1 = (sSlice + sLine) / 2;
    int sEq2 = (sSlice - sLine) / 2;

    // Thread Index (1D)
    int xThreadIdx = threadIdx.x;
    int yThreadIdx = threadIdx.y;

    // Block Width & Height
    int blockWidth = blockDim.x;
    int blockHeight = blockDim.y;

    // Thread Index (2D)
    int xIndex = blockIdx.x * blockWidth + xThreadIdx;
    int yIndex = blockIdx.y * blockHeight + yThreadIdx;

    // Thread Index Converted into 1D Index
    int index = (yIndex * N) + xIndex;

    T regTemp;

    if (xIndex < N / 2)
    {
        if (yIndex < N / 2)
        {
            regTemp = data[index];

            // First Quad
            data[index] = data[index + sEq1];

            // Third Quad
            data[index + sEq1] = regTemp;
        }
    }
    else
    {
        if (yIndex < N / 2)
        {
            regTemp = data[index];

            // Second Quad
            data[index] = data[index + sEq2];

            // Fourth Quad
            data[index + sEq2] = regTemp;
        }
    }
}

#endif // CUFFTSHIFT_2D_SINGLE_ARRAY_CU
