#include "hip/hip_runtime.h"
#ifndef CUFFTSHIFT_3D_CU
#define CUFFTSHIFT_3D_CU

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cutil_inline.h>

template <typename T>
__global__
void cufftShift_3D_slice_kernel(T* input, T* output, int N, int zIndex)
{
    // 3D Volume & 2D Slice & 1D Line
    int sLine = N;
    int sSlice = N * N;
    int sVolume = N * N * N;

    // Transformations Equations
    int sEq1 = (sVolume + sSlice + sLine) / 2;
    int sEq2 = (sVolume + sSlice - sLine) / 2;
    int sEq3 = (sVolume - sSlice + sLine) / 2;
    int sEq4 = (sVolume - sSlice - sLine) / 2;

    // Thread
    int xThreadIdx = threadIdx.x;
    int yThreadIdx = threadIdx.y;

    // Block Width & Height
    int blockWidth = blockDim.x;
    int blockHeight = blockDim.y;

    // Thread Index 2D
    int xIndex = blockIdx.x * blockWidth + xThreadIdx;
    int yIndex = blockIdx.y * blockHeight + yThreadIdx;

    // Thread Index Converted into 1D Index
    int index = (zIndex * sSlice) + (yIndex * sLine) + xIndex;

    if (zIndex < N / 2)
    {
        if (xIndex < N / 2)
        {
            if (yIndex < N / 2)
            {
                // First Quad
                output[index] = input[index + sEq1];
            }
            else
            {
                // Third Quad
                output[index] = input[index + sEq3];
            }
        }
        else
        {
            if (yIndex < N / 2)
            {
                // Second Quad
                output[index] = input[index + sEq2];
            }
            else
            {
                // Fourth Quad
                output[index] = input[index + sEq4];
            }
        }
    }

    else
    {
        if (xIndex < N / 2)
        {
            if (yIndex < N / 2)
            {
                // First Quad
                output[index] = input[index - sEq4];
            }
            else
            {
                // Third Quad
                output[index] = input[index - sEq2];
            }
        }
        else
        {
            if (yIndex < N / 2)
            {
                // Second Quad
                output[index] = input[index - sEq3];
            }
            else
            {
                // Fourth Quad
                output[index] = input[index - sEq1];
            }
        }
    }
}

template <typename T>
void cufftShift_3D_kernel(T* input, T* output, int N, dim3 block, dim3 grid)
{
    for (int i = 0; i < N; i++)
        cufftShift_3D_slice_kernel <<< grid, block >>> (input, output, N, i);
}

template
void cufftShift_3D_kernel <hipfftReal>
(hipfftReal* input, hipfftReal* output, int N, dim3 block, dim3 grid);

template
void cufftShift_3D_kernel <hipfftDoubleReal>
(hipfftDoubleReal* input, hipfftDoubleReal* output, int N, dim3 block, dim3 grid);

template
void cufftShift_3D_kernel <hipfftComplex>
(hipfftComplex* input, hipfftComplex* output, int N, dim3 block, dim3 grid);

template
void cufftShift_3D_kernel <hipfftDoubleComplex>
(hipfftDoubleComplex* input, hipfftDoubleComplex* output, int N, dim3 block, dim3 grid);

#endif // CUFFTSHIFT_3D_CU
