#include "hip/hip_runtime.h"
#ifndef CUFFTSHIFT_1D_CU
#define CUFFTSHIFT_1D_CU

#include <hip/hip_runtime.h>
#include <cutil_inline.h>

template <typename T>
__global__
void cufftShift_1D_kernel(T* input, T* output, int NX)
{
    int threadIdxX = threadIdx.x;
    int blockDimX = blockDim.x;
    int blockIdxX = blockIdx.x;

    int index = ((blockIdxX * blockDimX) + threadIdxX);

    if (index < (NX / 2))
        output[index] = (T) input[index + (NX / 2)];
    else
        output[index] = (T) input[index - (NX / 2)];
}


#endif // CUFFTSHIFT_1D_CU
