#include "hip/hip_runtime.h"
#ifndef CUFFTSHIFT_2D_CU
#define CUFFTSHIFT_2D_CU

#include <hip/hip_runtime.h>
#include <cutil_inline.h>

template <typename T>
__global__
void cufftShift_2D_kernel(T* input, T* output, int N)
{
    // 2D Slice & 1D Line
    int sLine = N;
    int sSlice = N * N;

    // Transformations Equations
    int sEq1 = (sSlice + sLine) / 2;
    int sEq2 = (sSlice - sLine) / 2;

    __syncthreads();

    // Thread Index (1D)
    int xThreadIdx = threadIdx.x;
    int yThreadIdx = threadIdx.y;

    __syncthreads();

    // Block Width & Height
    int blockWidth = blockDim.x;
    int blockHeight = blockDim.y;

    __syncthreads();

    // Thread Index (2D)
    int xIndex = blockIdx.x * blockWidth + xThreadIdx;
    int yIndex = blockIdx.y * blockHeight + yThreadIdx;

    __syncthreads();

    // Thread Index Converted into 1D Index
    int index = (yIndex * N) + xIndex;

    __syncthreads();

    if (xIndex < N / 2)
    {
        if (yIndex < N / 2)
        {
            // First Quad
            output[index] = input[index + sEq1];
            __syncthreads();
        }
        else
        {
            // Third Quad
            output[index] = input[index - sEq2];
            __syncthreads();
        }
    }
    else
    {
        if (yIndex < N / 2)
        {
            // Second Quad
            output[index] = input[index + sEq2];
            __syncthreads();
        }
        else
        {
            // Fourth Quad
            output[index] = input[index - sEq1];
            __syncthreads();
        }
    }
}

#endif // CUFFTSHIFT_2D_CU
