#ifndef CUFFTSHIFT_2D_SINGLE_ARRAY_IMPL_CU
#define CUFFTSHIFT_2D_SINGLE_ARRAY_IMPL_CU

#include "configGPU.h"
#include "cufftShiftShared.h"
#include "in-place/cufftShift_2D_IP.cu"

template <typename T>
extern
void cufftShift_2D_impl(T* data, int NX, int NY)
{
    if (NX == NY)
    {
        const int N = NX;
        kernelConf* conf = cufftShift::GenAutoConf_2D(N/2);
        cufftShift_2D_kernel <<< conf->grid, conf->block >>> (data, N);
    }
    else
    {
        printf("The library is supporting NxN arrays only \n");
        exit(0);
    }
}

template <typename T>
extern
void cufftShift_2D_config_impl(T* data, int NX, int NY, kernelConf* conf)
{
    if (NX == NY)
    {
        const int N = NX;
        cufftShift_2D_kernel <<< conf->grid, conf->block >>> (data, N);
    }

    else
    {
        printf("The library is supporting NxN arrays only \n");
        exit(0);
    }
}

template void cufftShift_2D_impl <hipfftReal>
(hipfftReal* data, int NX, int NY);

template void cufftShift_2D_impl <hipfftDoubleReal>
(hipfftDoubleReal* data, int NX, int NY);

template void cufftShift_2D_impl <hipfftComplex>
(hipfftComplex* data, int NX, int NY);

template void cufftShift_2D_impl <hipfftDoubleComplex>
(hipfftDoubleComplex* data, int NX, int NY);

template void cufftShift_2D_config_impl <hipfftReal>
(hipfftReal* data, int NX, int NY, kernelConf* conf);

template void cufftShift_2D_config_impl <hipfftDoubleReal>
(hipfftDoubleReal* data, int NX, int NY, kernelConf* conf);

template void cufftShift_2D_config_impl <hipfftComplex>
(hipfftComplex* data, int NX, int NY, kernelConf* conf);

template void cufftShift_2D_config_impl <hipfftDoubleComplex>
(hipfftDoubleComplex* data, int NX, int NY, kernelConf* conf);

#endif // CUFFTSHIFT_2D_SINGLE_ARRAY_IMPL_CU
