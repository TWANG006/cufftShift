#ifndef CUFFTSHIFT_1D_SINGLE_ARRAY_IMPL_CU
#define CUFFTSHIFT_1D_SINGLE_ARRAY_IMPL_CU

#include "configGPU.h"
#include "cufftShiftShared.h"
#include "in-place/cufftShift_1D_IP.cu"

template <typename T>
extern
void cufftShift_1D_impl(T* data, int NX)
{
    kernelConf* conf = cufftShift::GenAutoConf_1D(NX/2);
    cufftShift_1D_kernel <<< conf->grid, conf->block >>> (data, NX);
}

template <typename T>
extern
void cufftShift_1D_config_impl(T* data, int NX, kernelConf* conf)
{
    cufftShift_1D_kernel <<< conf->grid, conf->block >>> (data, NX);
}

template void cufftShift_1D_impl <hipfftReal>
(hipfftReal* data, int NX);

template void cufftShift_1D_impl <hipfftDoubleReal>
(hipfftDoubleReal* data, int NX);

template void cufftShift_1D_impl <hipfftComplex>
(hipfftComplex* data, int NX);

template void cufftShift_1D_impl <hipfftDoubleComplex>
(hipfftDoubleComplex* data, int NX);

template void cufftShift_1D_config_impl <hipfftReal>
(hipfftReal* data, int NX, kernelConf* conf);

template void cufftShift_1D_config_impl <hipfftDoubleReal>
(hipfftDoubleReal* data, int NX, kernelConf* conf);

template void cufftShift_1D_config_impl <hipfftComplex>
(hipfftComplex* data, int NX, kernelConf* conf);

template void cufftShift_1D_config_impl <hipfftDoubleComplex>
(hipfftDoubleComplex* data, int NX, kernelConf* conf);

#endif // CUFFTSHIFT_1D_SINGLE_ARRAY_IMPL_CU
