#ifndef CUFFTSHIFT_3D_SINGLE_ARRAY_IMPL_CU
#define CUFFTSHIFT_3D_SINGLE_ARRAY_IMPL_CU

#include "configGPU.h"
#include "cufftShiftShared.h"
#include "in-place/cufftShift_3D_IP.cu"

template <typename T>
extern
void cufftShift_3D_impl(T* data, int NX, int NY, int NZ)
{
    if (NX == NY && NX == NZ)
    {
        const int N = NX;
        kernelConf* conf = cufftShift::GenAutoConf_3D(N/2);
        cufftShift_3D_kernel(data, N, conf->block, conf->grid);
    }
    else
    {
        printf("The library is supporting NxNxN arrays only \n");
        exit(0);
    }
}

template <typename T>
extern
void cufftShift_3D_config_impl(T* data, int NX, int NY, int NZ, kernelConf* conf)
{
    if (NX == NY && NX == NZ)
    {
        const int N = NX;
        cufftShift_3D_kernel(data, N, conf->block, conf->grid);
    }
    else
    {
        printf("The library is supporting NxNxN arrays only \n");
        exit(0);
    }
}

template void cufftShift_3D_impl <hipfftReal>
(hipfftReal* data, int NX, int NY, int NZ);

template void cufftShift_3D_impl <hipfftDoubleReal>
(hipfftDoubleReal* data, int NX, int NY, int NZ);

template void cufftShift_3D_impl <hipfftComplex>
(hipfftComplex* data, int NX, int NY, int NZ);

template void cufftShift_3D_impl <hipfftDoubleComplex>
(hipfftDoubleComplex* data, int NX, int NY, int NZ);

template void cufftShift_3D_config_impl <hipfftReal>
(hipfftReal* data, int NX, int NY, int NZ, kernelConf* conf);

template void cufftShift_3D_config_impl <hipfftDoubleReal>
(hipfftDoubleReal* data, int NX, int NY, int NZ, kernelConf* conf);

template void cufftShift_3D_config_impl <hipfftComplex>
(hipfftComplex* data, int NX, int NY, int NZ, kernelConf* conf);

template void cufftShift_3D_config_impl <hipfftDoubleComplex>
(hipfftDoubleComplex* data, int NX, int NY, int NZ, kernelConf* conf);

#endif // CUFFTSHIFT_3D_SINGLE_ARRAY_IMPL_CU
