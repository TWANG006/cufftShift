#ifndef CUFFTSHIFT_2D_IMPL_CU
#define CUFFTSHIFT_2D_IMPL_CU

#include "configGPU.h"
#include "cufftShiftShared.h"
#include "out-of-place/cufftShift_2D_OP.cu"

template <typename T>
extern
void cufftShift_2D_impl(T* input, T* output, int NX, int NY)
{
    if (NX == NY)
    {
        const int N = NX;
        kernelConf* conf = cufftShift::GenAutoConf_2D(N);
        cufftShift_2D_kernel <<< conf->grid, conf->block >>> (input, output, N);
    }
    else
    {
        printf("The library is supporting NxN arrays only \n");
        exit(0);
    }
}

template <typename T>
extern
void cufftShift_2D_config_impl(T* input, T* output, int NX, int NY, kernelConf* conf)
{
    if (NX == NY)
    {
        const int N = NX;
        cufftShift_2D_kernel <<< conf->grid, conf->block >>> (input, output, N);
    }

    else
    {
        printf("The library is supporting NxN arrays only \n");
        exit(0);
    }
}

template void cufftShift_2D_impl <hipfftReal>
(hipfftReal* input, hipfftReal* output, int NX, int NY);

template void cufftShift_2D_impl <hipfftDoubleReal>
(hipfftDoubleReal* input, hipfftDoubleReal* output, int NX, int NY);

template void cufftShift_2D_impl <hipfftComplex>
(hipfftComplex* input, hipfftComplex* output, int NX, int NY);

template void cufftShift_2D_impl <hipfftDoubleComplex>
(hipfftDoubleComplex* input, hipfftDoubleComplex* output, int NX, int NY);

template void cufftShift_2D_config_impl <hipfftReal>
(hipfftReal* input, hipfftReal* output, int NX, int NY, kernelConf* conf);

template void cufftShift_2D_config_impl <hipfftDoubleReal>
(hipfftDoubleReal* input, hipfftDoubleReal* output, int NX, int NY, kernelConf* conf);

template void cufftShift_2D_config_impl <hipfftComplex>
(hipfftComplex* input, hipfftComplex* output, int NX, int NY, kernelConf* conf);

template void cufftShift_2D_config_impl <hipfftDoubleComplex>
(hipfftDoubleComplex* input, hipfftDoubleComplex* output, int NX, int NY, kernelConf* conf);

#endif // CUFFTSHIFT_2D_IMPL_CU
