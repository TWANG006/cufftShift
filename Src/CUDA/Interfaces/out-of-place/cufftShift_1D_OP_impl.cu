#ifndef CUFFTSHIFT_1D_IMPL_CU
#define CUFFTSHIFT_1D_IMPL_CU

#include "configGPU.h"
#include "cufftShiftShared.h"
#include "out-of-place/cufftShift_1D_OP.cu"

template <typename T>
extern
void cufftShift_1D_impl(T* input, T* output, int NX)
{
    kernelConf* conf = cufftShift::GenAutoConf_1D(NX);
    cufftShift_1D_kernel <<< conf->grid, conf->block >>> (input, output, NX);
}

template <typename T>
extern
void cufftShift_1D_config_impl(T* input, T* output, int NX, kernelConf* conf)
{
    cufftShift_1D_kernel <<< conf->grid, conf->block >>> (input, output, NX);
}

template void cufftShift_1D_impl <hipfftReal>
(hipfftReal* input, hipfftReal* output, int NX);

template void cufftShift_1D_impl <hipfftDoubleReal>
(hipfftDoubleReal* input, hipfftDoubleReal* output, int NX);

template void cufftShift_1D_impl <hipfftComplex>
(hipfftComplex* input, hipfftComplex* output, int NX);

template void cufftShift_1D_impl <hipfftDoubleComplex>
(hipfftDoubleComplex* input, hipfftDoubleComplex* output, int NX);

template void cufftShift_1D_config_impl <hipfftReal>
(hipfftReal* input, hipfftReal* output, int NX, kernelConf* conf);

template void cufftShift_1D_config_impl <hipfftDoubleReal>
(hipfftDoubleReal* input, hipfftDoubleReal* output, int NX, kernelConf* conf);

template void cufftShift_1D_config_impl <hipfftComplex>
(hipfftComplex* input, hipfftComplex* output, int NX, kernelConf* conf);

template void cufftShift_1D_config_impl <hipfftDoubleComplex>
(hipfftDoubleComplex* input, hipfftDoubleComplex* output, int NX, kernelConf* conf);

#endif // CUFFTSHIFT_1D_IMPL_CU
