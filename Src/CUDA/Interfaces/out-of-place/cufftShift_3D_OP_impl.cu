#ifndef CUFFTSHIFT_3D_IMPL_CU
#define CUFFTSHIFT_3D_IMPL_CU

#include "configGPU.h"
#include "cufftShiftShared.h"
#include "out-of-place/cufftShift_3D_OP.cu"

template <typename T>
extern
void cufftShift_3D_impl(T* input, T* output, int NX, int NY, int NZ)
{
    if (NX == NY && NX == NZ)
    {
        const int N = NX;
        kernelConf* conf = cufftShift::GenAutoConf_3D(N);
        cufftShift_3D_kernel(input, output, N, conf->block, conf->grid);
    }
    else
    {
        printf("The library is supporting NxNxN arrays only \n");
        exit(0);
    }
}

template <typename T>
extern
void cufftShift_3D_config_impl(T* input, T* output, int NX, int NY, int NZ, kernelConf* conf)
{
    if (NX == NY && NX == NZ)
    {
        const int N = NX;
        cufftShift_3D_kernel(input, output, N, conf->block, conf->grid);
    }
    else
    {
        printf("The library is supporting NxNxN arrays only \n");
        exit(0);
    }
}

template void cufftShift_3D_impl <hipfftReal>
(hipfftReal* input, hipfftReal* output, int NX, int NY, int NZ);

template void cufftShift_3D_impl <hipfftDoubleReal>
(hipfftDoubleReal* input, hipfftDoubleReal* output, int NX, int NY, int NZ);

template void cufftShift_3D_impl <hipfftComplex>
(hipfftComplex* input, hipfftComplex* output, int NX, int NY, int NZ);

template void cufftShift_3D_impl <hipfftDoubleComplex>
(hipfftDoubleComplex* input, hipfftDoubleComplex* output, int NX, int NY, int NZ);

template void cufftShift_3D_config_impl <hipfftReal>
(hipfftReal* input, hipfftReal* output, int NX, int NY, int NZ, kernelConf* conf);

template void cufftShift_3D_config_impl <hipfftDoubleReal>
(hipfftDoubleReal* input, hipfftDoubleReal* output, int NX, int NY, int NZ, kernelConf* conf);

template void cufftShift_3D_config_impl <hipfftComplex>
(hipfftComplex* input, hipfftComplex* output, int NX, int NY, int NZ, kernelConf* conf);

template void cufftShift_3D_config_impl <hipfftDoubleComplex>
(hipfftDoubleComplex* input, hipfftDoubleComplex* output, int NX, int NY, int NZ, kernelConf* conf);

#endif // CUFFTSHIFT_3D_IMPL_CU
